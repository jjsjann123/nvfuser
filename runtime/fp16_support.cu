#include "hip/hip_runtime.h"

#define __NVFUSER_HALF_TO_US(var) *(reinterpret_cast<unsigned short*>(&(var)))
#define __NVFUSER_HALF_TO_CUS(var) \
  *(reinterpret_cast<const unsigned short*>(&(var)))

struct __half;
__device__ __inline__ __half __float2half(const float);

struct __align__(2) __half {
  __half() = default;

  __device__ __half(const float f) {
    __x = __float2half(f).__x;
  }

 protected:
  unsigned short __x;
};

__device__ __inline__ __half __float2half(const float f) {
  __half val;
  asm("{  cvt.rn.f16.f32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "f"(f));
  return val;
}

__device__ __inline__ __half __double2half(const double d) {
#if __CUDA_ARCH__ >= 900
  __half val;
  asm("{  cvt.rn.f16.f64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "d"(d));
  return val;
#else
  return __float2half(static_cast<float>(d));
#endif
}

__device__ __inline__ __half __int322half(const int i) {
#if __CUDA_ARCH__ >= 900
  __half val;
  asm("{  cvt.rn.f16.s32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "r"(i));
  return val;
#else
  return __float2half(static_cast<float>(i));
#endif
}

__device__ __inline__ __half __int2half(const int64_t i64) {
#if __CUDA_ARCH__ >= 900
  __half val;
  asm("{  cvt.rn.f16.s64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "l"(i64));
  return val;
#else
  return __float2half(static_cast<float>(i64));
#endif
}

__device__ __inline__ __half __bool2half(const bool b) {
  return __int2half((int)b);
}

__device__ __inline__ float __half2float(const __half h) {
  float val;
  asm("{  cvt.f32.f16 %0, %1;}\n" : "=f"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ double __half2double(const __half h) {
#if __CUDA_ARCH__ >= 900
  double val;
  asm("{  cvt.f64.f16 %0, %1;}\n" : "=d"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
#else
  return static_cast<double>(__half2float(h));
#endif
}

__device__ int __half2int32(const __half h) {
#if __CUDA_ARCH__ >= 900
  int val;
  asm("{  cvt.rzi.s32.f16 %0, %1;}\n"
      : "=r"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
#else
  return static_cast<int>(__half2float(h));
#endif
}

__device__ __inline__ int64_t __half2int(const __half h) {
#if __CUDA_ARCH__ >= 900
  int64_t val;
  asm("{  cvt.rzi.s64.f16 %0, %1;}\n"
      : "=l"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
#else
  return static_cast<int64_t>(__half2float(h));
#endif
}

__device__ __inline__ bool __half2bool(const __half h) {
  return (bool)__half2float(h) != 0;
}

__device__ __inline__ __half __real_then_2half(const std::complex<float> c) {
  return __float2half(std::real(c));
}

__device__ __inline__ __half __real_then_2half(const std::complex<double> c) {
  return __double2half(std::real(c));
}
